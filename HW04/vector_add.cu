#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(const float *A, const float *B, float *C int N) {
    int i = blockIDx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i]
    }
}

int main() {
    int N = 1000000;
    size_t size = N * sizeof(float);
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
}